#include "hip/hip_runtime.h"
///The following functions transforms an RGB image into a Gray scale image using CUDA

void __global__ rgb2gray(const float *pfimagIn, float *pfimgOut, const int iWidth, const int iHeight);

void ReadAndProcessImage();

#define BLOCK_SIZE 16

int main()
{
  //Here we call the rgb2gray function
  ReadAndProcessImage();
  
  std::cin.get();

}

void ReadAndProcessImage()
{
  int iWidth = 256;
  int iHeight = 256;
  
  float *h_ImagIn = new float[iWidth * iHeight];
  float *h_ImagOut = new float[iWidth * iHeight];
  
  //I need to get the values of the input filled with something
  
  float *d_ImagIn;
  float *d_imagOut;
  
  //Allocate memory in GPU
  hipMalloc((void **) &d_ImagIn, iWidth * iHeight * sizeof(float));
  hipMalloc((void **) &d_ImagOut, iWidth * iHeight * sizeof(float));
  
  //Transfer data to GPU
  hipMemcpy((void *) &d_ImagIn, (void *) &h_ImagIn, iWidth * iHeight * sizeof(float), hipMemcpyHostToDevice);
  
  //Compue the results in GPU
  Dim3 dimBlocks(BLOCK_SIZE, BLOCK_SIZE); //  Each thread block contains this much threads
  Dim3 dimGrid(iWidth / BLOCK_SIZE, iHeight / BLOCK_SIZE); // This amount of thread blocks
  //Total number of threads that will be launched are dimGrid.x * dimGird.y * dimBlocks.x * dimBlocks.y
  //NOTE: the toal numer of thread per block, i.e. dimBlock.x * dimBlock.y should not excede 1024 and
  //in some system 512
  
  rgb2gray <<< dimGrid, dimBlocks>>>  (d_ImagIn, d_ImgOut, iWidth, iHeight);
  
  //Transfer data back from GPU to CPU
  hipMemcpy((void *) &h_ImagOut, (void *) &d_ImagOut, iWidth * iHeight * sizeof(float), hipMemcpyDeviceToHost);
  
  delete pfImagIn;
  delete pfImagOut;
  
  hipFree(d_ImagIn);
  hipFree(d_ImagOut);
}

//GPU Kernel
void __global__ rgb2gray(const float *pfimagIn, float *pfimgOut, const int iWidth, const int iHeight)
{
  
  int iRow = blockIdx.y * blockDim.y + threadIdx.y;
  int iCol = blockIdx.x + blockDim.x + threadIdx.x;

  float fPixel = pfImagIn[iRow * iWidth + iCol];
  fPixel = fPixel & (0x000000FF); // Just get the R value; image format is ARGB

  float fPixelOut = fPixel & (fPixel << 8) & (fPixel << 16); //Gray scale so R = G = B
  fPixelOut = fPixelOut && 0x00000000; //No Transparency in Alpha channel;
  
  pfImageOut[iRow * iWidth + iCol] = fPixelOut;
  

}

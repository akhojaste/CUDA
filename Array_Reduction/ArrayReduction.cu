#include "hip/hip_runtime.h"
// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes CUDA
#include <hip/hip_runtime.h>

// includes, project
#include <hip/hip_runtime_api.h>
#include <helper_functions.h> // helper functions for SDK examples


__global__ void Array_Reduction_Kernel(float * g_iData, float *g_Intermediat)
{
	extern __shared__ float sData[]; //Size is determined by the host

	int index = blockDim.x * blockIdx.x + threadIdx.x;
	int tId = threadIdx.x;

	//First every thread in the block puts its value intor the shared memory
	sData[tId] = g_iData[index];
	__syncthreads();

	//VERSION 1
	//This mod makes it inefficient
	//for (int t = 1; t < blockDim.x; t *= 2)
	//{
	//	if (tId % (2 * t) == 0) //Even threads or even elements of the array. i.e. thread0, thread2, ...
	//	{
	//		sData[tId] += sData[tId + t];
	//	}
	//	//At each iteration of the for loop. Thread wait at this barrier so other threads will read here.
	//	__syncthreads();
	//}

	//VERSION 2, this has bank conflict problem
	//	//Bank conflict means different threads are accessing
	//	//adjacent values in the memory. How far the threads can access
	//	//depends on the Bank Size. For example, if the bank size is 4 bytes
	//	//then the memory access would be as follow:

	//	//Bank    |      1     |      2     |      3     | ...
	//	//Address | 0  1  2  3 | 4  5  6  7 | 8  9 10 11 | ...
	//	//Address | 64 65 66 67 | 68 69 70 71 | 72 73 74 75 | ...

	//	//Now if two threads aceess the adderesses 0, 1, this is inefficient
	//	//cause GPU will serialize these two threads. They have to be on two different banks
	//	//so we can aceess the memory in parallel.

	//for (int t = 1; t < blockDim.x; t *= 2)
	//{
	//	//We should do this
	//	int index2 = 2 * tId * t;
	//	if (index2 < blockDim.x)
	//	{
	//		sData[index2] += sData[index2 + t];
	//	}
	//	//At each iteration of the for loop. Thread wait at this barrier so other threads will read here.
	//	__syncthreads();
	//}

	//VERSION 3
	//Fixing the bank conflicts. Make the threads to acess further threads.
	for (unsigned int idx = blockDim.x / 2; idx > 0; idx >>= 1)
	{
		if (tId < idx)
		{
			sData[tId] += sData[idx + tId];
		}
		
		__syncthreads();
	}

	//Move the summation to the global memory now
	if (tId == 0)
	{
		g_Intermediat[blockIdx.x] = sData[0];
	}
}

extern "C"
void Array_Reduction(float *h_ArrayReduction, unsigned int ArraySize, long &lSum)
{
	int blocks = 1024;
	int threadPerBlock = 1024; // 1024 blocks of 1024 threads per each block
	int iMemSize = threadPerBlock * sizeof(float); // to pass to the cuda kernel so it allocates shared memory there

	//Device array
	float *d_ArrayReduction = nullptr;
	checkCudaErrors(hipMalloc((void **)&d_ArrayReduction, ArraySize * sizeof(float)));

	float *d_ArrayReductionIntermediate = nullptr;
	checkCudaErrors(hipMalloc((void **)&d_ArrayReductionIntermediate, blocks * sizeof(float)));

	float *d_ArrayReductionOut = nullptr;
	checkCudaErrors(hipMalloc((void **)&d_ArrayReductionOut, 1 * sizeof(float)));

	//Move the data from host to device
	checkCudaErrors(hipMemcpy(d_ArrayReduction, h_ArrayReduction, ArraySize * sizeof(float), hipMemcpyHostToDevice));

	long long lStart, lEnd, lFreq;
	QueryPerformanceFrequency((LARGE_INTEGER*)&lFreq);

	QueryPerformanceCounter((LARGE_INTEGER*)&lStart);
	
	//First call, each block will compute its summation and will put it in one element of output array
	//So we get one array of 1024 elements.
	Array_Reduction_Kernel << <blocks, threadPerBlock, iMemSize >> >(d_ArrayReduction, d_ArrayReductionIntermediate);

	//Second call, we have 1024 elemets left, with 1 block of 1024 elemnts each, we compute the summation.
	blocks = 1;
	threadPerBlock = 1024;
	Array_Reduction_Kernel <<<blocks, threadPerBlock, iMemSize >>> (d_ArrayReductionIntermediate, d_ArrayReductionOut);

	QueryPerformanceCounter((LARGE_INTEGER*)&lEnd);
	double dbTime = (lEnd - lStart)* 1000;
	dbTime /= lFreq;

	std::cout << "Total time: " << dbTime << " ms" << std::endl;

	checkCudaErrors(hipMemcpy(h_ArrayReduction, d_ArrayReductionOut, 1 * sizeof(float), hipMemcpyDeviceToHost));

	lSum = h_ArrayReduction[0];

	checkCudaErrors(hipFree(d_ArrayReduction));
}
